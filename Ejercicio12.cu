
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
#define BLOCK_SIZE 32
typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float Cvalue = 0;

    int row = threadIdx.y;
    int col = threadIdx.x;

    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        __syncthreads();
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        __syncthreads();
    }

    SetElement(Csub, row, col, Cvalue);
}


int main() {
    cout<<  "inicie" << endl;
    int s = 64;
    int size = s * s * sizeof(float);
    float * inputCPU = NULL;
    inputCPU = (float*) malloc (size);
    for (int j=0;j<(s*s);j++) {
        inputCPU[j]=j+1.1f;
    }

   
   Matrix A;
   A.width = s;
   A.height = s;
   A.stride = 1;
   A.elements = inputCPU;


      Matrix B;
   B.width = s;
   B.height = s;
   B.stride = 1;
   B.elements = inputCPU;

      Matrix C;
   C.width = s;
   C.height = s;
   C.stride = 1;
   C.elements = inputCPU;
   cout<<  "mult" << endl;
   MatMul(A,  B,C);
   cout<<  "mC" << endl;
   for (int j=0;j<(s*s);j++) {
       cout<<  C.elements[j]<< endl;
    }


  return 0;
}
